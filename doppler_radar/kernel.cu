#include "hip/hip_runtime.h"
﻿
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "kernel.h"


void __global__  fftshift(Complex*, int);

void my_CUFFT(json& data, size_t new_size, json& outPutJson) {


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const size_t mem_size = sizeof(Complex) * new_size;


	std::cout << "CUFFT is starting..." << std::endl;


	// Allocate host memory for the signal
	Complex* h_signal =
		reinterpret_cast<Complex*>(malloc(sizeof(Complex) * new_size));

	Complex* h_outPut =
		reinterpret_cast<Complex*>(malloc(sizeof(Complex) * new_size));



	// Initialize the memory for the signal
	for (unsigned int i = 0; i < new_size; ++i) {
		h_signal[i].x = data["real"][i];

		h_signal[i].y = data["imag"][i];
	}




	// Allocate device memory for signal
	Complex* d_signal;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), mem_size));
	//checkCudaErrors(hipMalloc(&d_signal, mem_size));

	Complex* d_outPut;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_outPut), mem_size));

	hipEventRecord(start);

	// Copy host memory to device
	checkCudaErrors(
		hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

	// CUFFT plan 
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

	//hipEventRecord(start);


	// Transform signal and kernel
	std::cout << "Transforming signal hipfftExecC2C" << std::endl;
	checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex*>(d_signal),
		reinterpret_cast<hipfftComplex*>(d_outPut),
		HIPFFT_FORWARD));

	/*hipDeviceSynchronize();*/
	// Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan));

	const int block_size=256;

	const int grid_size = (new_size + block_size - 1)/block_size;


	fftshift << <grid_size, block_size >> > (d_outPut, new_size);


	getLastCudaError("Kernel execution failed [ fftshift ]");
	//hipEventRecord(stop);


	checkCudaErrors(
		hipMemcpy(h_outPut, d_outPut, mem_size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("cufft和fftshift共耗时 %f 毫秒", milliseconds);

	hipDeviceSynchronize();

	//-----------------------------------------------------------------------------------


	for (unsigned int i = 0; i < new_size; ++i) {

		outPutJson["magnitude"].push_back(my_getMagnitude(h_outPut[i].x, h_outPut[i].y, 20));

	}


	//my_fftShift1D(outPutJson["magnitude"], new_size);
	//-----------------------------------------------------------------------------------

	free(h_signal);
	free(h_outPut);
	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_outPut));



}


void my_fftShift1D(json& myData, size_t N)
{
	const unsigned int halfLength = N / 2;
	float tempFloat;



	for (size_t i = 0; i < N; i++)
	{
		if (i < halfLength)
		{
			tempFloat = myData[i];
			myData[i] = myData[halfLength + i];
			myData[halfLength + i] = tempFloat;


		}

	}
}

float my_getMagnitude(float x, float y, int scale) {

	return scale * log10f(sqrtf(x * x + y * y));
}

static void __global__  fftshift(Complex* data, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	Complex temp;

	if (i < N / 2)
	{
		temp.x = data[i].x;
		temp.y = data[i].y;

		data[i].x = data[i + N / 2].x;
		data[i].y = data[i + N / 2].y;

		data[i + N / 2].x = temp.x;
		data[i + N / 2].y = temp.y;
	}
}
